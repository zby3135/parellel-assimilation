//parallel program

#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <cstdlib>
#include <ctime>
#include <vector>
#include <algorithm>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#define K 401296
#define max(a, b)  (((a) > (b)) ? (a) : (b))
#define min(a, b)  (((a) < (b)) ? (a) : (b))
#define NT 4
#define PS  20
#define PS2 10
#define Y 365
#define EP 15
#define CHECK(res) if(res!=hipSuccess){printf("Error:%d\n", __LINE__);exit(-1);}

__device__ double LIMIT(double min, double max, double X)
{
	if (min>X)
		return min;
	if (max<X)
		return max;
	return X;
}

__device__ double AFGEN(double *x, int n, double X)
{
	double Y1 = 0;
	if (X <= x[0])
		Y1 = x[1];
	if (X >= x[n - 2])
		Y1 = x[n - 1];
	for (int i = 2; i<n - 1;)
	{
		if ((x[i] >= X) && (X >= x[i - 2]))
		{
			double slope = (x[i + 1] - x[i - 1]) / (x[i] - x[i - 2]);
			Y1 = x[i - 1] + (X - x[i - 2])*slope;
		}
		i += 2;
	}
	return Y1;
}

__device__ double sum(double *f, int k)
{
	double sum = 0;
	for (int i = 0; i<k; i++)
	{
		if (f[i]>0)
			sum = sum + f[i];
	}
	return sum;
}

__device__ double max2(double *f, int k)
{
	double max2 = 0;
	for (int i = 0; i<k - 1; i++)
	{
		if (f[i + 1]>f[i])
			max2 = f[i + 1];
	}
	return max2;
}

//WOFOST model
__device__ void LAIcal(double SPAN, double *param, double *TMIN, double *TMAX, double *AVRAD, double *JCLAImoni)
{
	double IDEM = param[0];	
	int IDAY = 0;       
	int DELT = 1;	
	double DTSMTB[] = { 0.00, 0.00, 10.00, 0.00, 30.00, 20.00, 40.00, 30.00 };
	int ILDTSM = sizeof(DTSMTB) / sizeof(double);
	double TSUM1 = 1800;         
	double TSUM2 = 620;          
	double DVSI = 0.250;         
	double DVSEND = 2.00;          	
	double TDWI = 65.00;        
	double RGRLAI = 0.0070;      
	double SLATB[] = { 0.00, 0.0045, 0.16, 0.0033, 0.61, 0.0030, 0.80, 0.0029, 1.00, 0.0025, 1.55, 0.0024, 2.02, 0.0018 };
	int ILSLA = sizeof(SLATB) / sizeof(double);	
	double LAIEM = 0.10;          	
	double TBASE = 15.0;          	
	double KDIFTB[] = { 0.00, 0.40, 0.65, 0.40, 1.00, 0.60, 2.00, 0.60 };
	int ILKDIF = sizeof(KDIFTB) / sizeof(double);	
	double EFFTB[] = { 10, 0.54, 40, 0.36 };
	int ILEFF = sizeof(EFFTB) / sizeof(double);	
	double AMAXTB[] = { 0.00, 40.00, 1.00, 40, 1.90, 40, 2.00, 40.00 };
	int ILAMAX = sizeof(AMAXTB) / sizeof(double);	
	double TMPFTB[] = { 0.00, 0.00, 12.00, 0.69, 18.00, 0.85, 24.00, 1.00, 30.00, 1.00, 36.00, 0.87, 42.00, 0.27 };
	int ILTMPF = sizeof(TMPFTB) / sizeof(double);
	double TMNFTB[] = { 0.00, 0.00, 3.00, 1.00 };
	int ILTMNF = sizeof(TMNFTB) / sizeof(double);
	double CVL = 0.754;        
	double CVO = 0.684;       
	double CVR = 0.754;       
	double CVS = 0.754;       
	double Q10 = 2.0;     
	double RML = 0.0200; 
	double RMO = 0.0030; 
	double RMR = 0.0100; 
	double RMS = 0.0150; 
	double RFSETB[] = { 0.00, 1.00, 2.00, 1.00 };
	int ILRFSE = sizeof(RFSETB) / sizeof(double);
	double FLTB[] = { 0.00, 0.65, 0.31, 0.60, 0.53, 0.57, 0.80, 0.35, 0.94, 0.14, 1.00, 0.10, 1.2, 0.00, 2.10, 0.00 };
	int ILFL = sizeof(FLTB) / sizeof(double);
	double FRTB[] = { 0.00, 0.50, 0.43, 0.45, 0.65, 0.40, 0.80, 0.37, 0.85, 0.27, 0.99, 0.10, 1.00, 0.00, 2.00, 0.00 };
	int ILFR = sizeof(FRTB) / sizeof(double);
	double FSTB[] = { 0.00, 0.35, 0.31, 0.40, 0.53, 0.43, 0.80, 0.637, 0.94, 0.553, 1.00, 0.10, 1.20, 0.00, 2.10, 0.00 };
	int ILFS = sizeof(FSTB) / sizeof(double);
	double FOTB[] = { 0.00, 0.00, 0.50, 0.00, 0.80, 0.013, 0.94, 0.316, 1.00, 0.80, 1.20, 1.000, 1.50, 1.000, 2.00, 1.00 };
	int ILFO = sizeof(FOTB) / sizeof(double);
    double RDRRTB[] = { 0.00, 0.000, 1.50, 0.000, 1.5001, 0.020, 2.00, 0.020 };
	int ILRDRR = sizeof(RDRRTB) / sizeof(double);
	double RDRSTB[] = { 0.00, 0.000, 1.50, 0.000, 1.5001, 0.020, 2.00, 0.020 };
	int ILRDRS = sizeof(RDRSTB) / sizeof(double);
	double IDANTH = -99;       
	double DVS = DVSI;   
	double TSUM = 0;     
	double FR = AFGEN(FRTB, ILFR, DVS);
	double FL = AFGEN(FLTB, ILFL, DVS);
	double FS = AFGEN(FSTB, ILFS, DVS);
	double FO = AFGEN(FOTB, ILFO, DVS);
	double SLA[Y]; SLA[0] = AFGEN(SLATB, ILSLA, DVS);
	double LVAGE[Y]; LVAGE[0] = 0;             
	double ILVOLD = 1;                        
	double WRT = FR*TDWI;        
	double TADW = (1 - FR)*TDWI;  
	double WST = FS*TADW;        
	double WSO = FO*TADW;        
	double WLV = FL*TADW;        
	LAIEM = WLV * SLA[0];        
	double LV[Y]; LV[0] = WLV;	
	double LASUM = LAIEM;      
	double LAIEXP = LAIEM;     
	double LAIMAX = LAIEM;     
	double LAI[Y]; LAI[int(IDEM) - 1] = LASUM;    
	double TMINRA = 0;
	double TMNSAV[7];
	for (int i1 = 0; i1<7; i1++)
	{
		TMNSAV[i1] = -99;
	}
	double TEMP;
	double DTEMP;
	double GASS;
	double MRES;
	double DMI;
	double DTSUM;
	double DVR;
	double AMAX;
	double KDIF;
	double EFF;
	double XGAUSS[3] = { 0.1127017, 0.5000000, 0.8872983 };
	double WGAUSS[3] = { 0.2777778, 0.4444444, 0.2777778 };
	double PI = 3.1415926;
	double DALV;
	for (IDAY = IDEM - 1; IDAY < Y; IDAY++)
	{
		TEMP = (TMIN[IDAY] + TMAX[IDAY]) / 2;      
		DTEMP = (TMAX[IDAY] + TEMP) / 2;       
		for (int i2 = 0; i2 < 6; i2++)
			TMNSAV[i2] = TMNSAV[i2 + 1];
		TMNSAV[6] = TMIN[IDAY];
		TMINRA = 0;
		int I4 = 0;
		for (int i3 = 0; i3 < 7; i3++)
		{
			if (TMNSAV[i3] != -99)
			{
				TMINRA = TMINRA + TMNSAV[i3];
				I4 = I4 + 1;
			}
		}
		TMINRA = TMINRA / I4;
		DTSUM = AFGEN(DTSMTB, ILDTSM, TEMP);
		if (DVS < 1)
			DVR = DTSUM / TSUM1;   
		else
			DVR = DTSUM / TSUM2;   
		AMAX = AFGEN(AMAXTB, ILAMAX, DVS);
		AMAX = AMAX * AFGEN(TMPFTB, ILTMPF, DTEMP);
		KDIF = AFGEN(KDIFTB, ILKDIF, DVS);
		EFF = AFGEN(EFFTB, ILEFF, DTEMP);
		double DTGA = 0;
		double DAYL;
		for (int i5 = 0; i5 < 3; i5++)
		{
			double DEC = -asin(sin(23.45*0.0174533)*cos(2 * PI*(IDAY + 1 + 10) / Y));
			double SINLD = sin(0.017453292*43.85)*sin(DEC);
			double COSLD = cos(0.017453292*43.85)*cos(DEC);
			double AOB = SINLD / COSLD;
			DAYL = 12 * (1 + 2 * asin(AOB) / PI);           
			double HOUR = 12 + 0.5*DAYL*XGAUSS[i5];         
			double HOUR2 = SINLD + COSLD*cos(2 * PI*(HOUR + 12) / 24);
			double SINB = max(0, HOUR2);
			double DSINBE = 3600 * (DAYL*(SINLD + 0.4*(SINLD*SINLD + COSLD*COSLD*0.5)) + 12 * COSLD*(2 + 3 * 0.4*SINLD)*sqrt(1 - AOB*AOB) / PI);
			double PAR = 0.5*AVRAD[IDAY] * SINB*(1 + 0.4*SINB) / DSINBE;
			double SC = 1370 * (1 + 0.033*cos(2 * PI*(IDAY + 1) / Y));
			double DSINB = 3600 * (DAYL*SINLD + 24 * COSLD*sqrt(1 - AOB*AOB) / PI);
			double ANGOT = SC*DSINB;
			double ATMTR = AVRAD[IDAY] / ANGOT;
			double FRDIF;
			if (ATMTR > 0.75)
			{
				FRDIF = 0.23;
			}
			else
			{
				if (0.35 < ATMTR&ATMTR <= 0.75)
				{
					FRDIF = 1.33 - 1.46*ATMTR;
				}
				else
				{
					if (0.07 < ATMTR&ATMTR <= 0.35)
					{
						FRDIF = 1 - 2.3*(ATMTR - 0.07)*(ATMTR - 0.07);
					}
					else
						FRDIF = 1;
				}
			}
			double DIFPP = FRDIF*ATMTR*0.5*SC;
			double PARDIF = min(PAR, SINB*DIFPP);
			double PARDIR = PAR - PARDIF;
			double SCV = 0.2;									
			double REFH = (1 - sqrt(1 - SCV)) / (1 + sqrt(1 - SCV));
			double REFS = REFH * 2 / (1 + 1.6*SINB);				 
			double KDIRBL = (0.5 / SINB)*KDIF / (0.8*sqrt(1 - SCV));  
			double KDIRT = KDIRBL*sqrt(1 - SCV);                 
			double FGROS = 0;
			for (int i6 = 0; i6 < 3; i6++)
			{
				double LAIC = LAI[IDAY] * XGAUSS[i6];
				double VISDF = (1 - REFS)*PARDIF*KDIF*exp(-KDIF*LAIC);
				double VIST = (1 - REFS)*PARDIR*KDIRT*exp(-KDIRT*LAIC);
				double VISD = (1 - SCV)*PARDIR*KDIRBL*exp(-KDIRBL*LAIC);
				double VISSHD = VISDF + VIST - VISD;
				double FGRSH = AMAX*(1 - exp(-VISSHD*EFF / max(2.0, AMAX)));
				double VISPP = (1 - SCV)*PARDIR / SINB;
				double FGRSUN;
				if (VISPP <= 0)
					FGRSUN = FGRSH;
				else
					FGRSUN = AMAX*(1 - (AMAX - FGRSH)*(1 - exp(-VISPP*EFF / max(2.0, AMAX))) / (EFF*VISPP));

				double FSLLA = exp(-KDIRBL*LAIC);
				double FGL = FSLLA*FGRSUN + (1 - FSLLA)*FGRSH;
				FGROS = FGROS + FGL*WGAUSS[i6];
			}
			FGROS = FGROS*LAI[IDAY];
			DTGA = DTGA + FGROS*WGAUSS[i5];
		}
		DTGA = DTGA*DAYL;
		DTGA = DTGA * AFGEN(TMNFTB, ILTMNF, TMINRA); 
		GASS = DTGA * 30 / 44;    
		double RMRES = (RMR*WRT + RML*WLV + RMS*WST + RMO*WSO)* AFGEN(RFSETB, ILRFSE, DVS);
		double TEFF = pow(Q10, (TEMP - 25) / 10);   
		MRES = min(GASS, RMRES*TEFF);
		double ASRC = GASS - MRES;
		FR = AFGEN(FRTB, ILFR, DVS);
		FL = AFGEN(FLTB, ILFL, DVS);
		FS = AFGEN(FSTB, ILFS, DVS);
		FO = AFGEN(FOTB, ILFO, DVS);
		double CVF = 1 / ((FL / CVL + FS / CVS + FO / CVO)*(1 - FR) + FR / CVR); 
		DMI = CVF*ASRC;                   
		double GRRT = FR*DMI;                               
		double DRRT = WRT * AFGEN(RDRRTB, ILRDRR, DVS);
		double GWRT = GRRT - DRRT;       
		double ADMI = (1 - FR)*DMI;
		double GRST = FS*ADMI;                                  
		double DRST = AFGEN(RDRSTB, ILRDRS, DVS)*WST;
		double GWST = GRST - DRST;                   
		double GWSO = FO*ADMI;              
		double GRLV = FL*ADMI;									  
		double LAICR = 3.2 / KDIF;
		double DSLV = WLV * LIMIT(0, 0.03, 0.03*(LAI[IDAY] - LAICR) / LAICR);
		int I7 = ILVOLD - 1;                                 
		while (DSLV > LV[I7] & I7 >= 0)
		{
			DSLV = DSLV - LV[I7];
			I7 = I7 - 1;
		}
		DALV = 0;
		if (LVAGE[I7] > SPAN & DSLV > 0 & I7 >= 0)
		{
			DALV = LV[I7] - DSLV;
			DSLV = 0;
			I7 = I7 - 1;
		}
		while (I7 >= 0 & LVAGE[I7] > SPAN)
		{
			DALV = DALV + LV[I7];
			I7 = I7 - 1;
		}
		DALV = DALV / DELT;		
		double FYSDEL = max(0, (TEMP - TBASE) / (35 - TBASE)); 
		double SLAT = AFGEN(SLATB, ILSLA, DVS);
		double DTEFF;
		double GLAIEX;
		double GLASOL;
		double GLA;
		if (LAIEXP<6)
		{
			DTEFF = max(0, TEMP - TBASE);
			GLAIEX = LAIEXP*RGRLAI*DTEFF;
			GLASOL = GRLV*SLAT;
			GLA = min(GLAIEX, GLASOL);
			if (GRLV>0)
				SLAT = GLA / GRLV;
		}
		DVS = DVS + DVR*DELT;
		TSUM = TSUM + DTSUM*DELT;
		if (DVS >= 1 & IDANTH == -99)
		{
			IDANTH = IDAY - IDEM;
			DVS = 1;
		}
		double DSLVT = DSLV*DELT;
		int I8 = ILVOLD - 1;
		while (DSLVT > 0 & I8 >= 0)
		{
			if (DSLVT >= LV[I8])
			{
				DSLVT = DSLVT - LV[I8];
				LV[I8] = 0;
				I8 = I8 - 1;
			}
			else
			{
				LV[I8] = LV[I8] - DSLVT;
				DSLVT = 0;
			}
		}
		while (LVAGE[I8] >= SPAN&I8 >= 0)
		{
			LV[I8] = 0;
			I8 = I8 - 1;
		}
		ILVOLD = I8 + 1;
		int I9;
		for (I9 = ILVOLD - 1; I9 > -1; I9--)
		{
			LV[I9 + 1] = LV[I9];
			SLA[I9 + 1] = SLA[I9];
			LVAGE[I9 + 1] = LVAGE[I9] + FYSDEL*DELT;
		}
		ILVOLD = ILVOLD + 1;
		LV[0] = GRLV*DELT;
		SLA[0] = SLAT;
		LVAGE[0] = 0;
		LASUM = 0;
		double tt = 0;
		int I10;
		for (I10 = 0; I10 < ILVOLD; I10++)
			tt = tt + LV[I10] * SLA[I10];
		LASUM = tt;
		WLV = sum(LV, Y);
		LAIEXP = LAIEXP + GLAIEX*DELT;
		WRT = WRT + GWRT*DELT;   
		WST = WST + GWST*DELT;
		WSO = WSO + GWSO*DELT;
		TADW = WLV + WST + WSO;		
		LAI[IDAY + 1] = LASUM;
		LAIMAX = max(LAI[IDAY + 1], LAIMAX);
		if (ILVOLD > 364)
			break;
		if (DVS >= DVSEND)
			break;
		LAIMAX = max2(LAI, Y);
		if (LAIMAX <= 0.002 & DVS > 0.5)
			break;
	}	
	for (int i = 0; i < NT; i++)
	{		
		JCLAImoni[i]=LAI[int(param[i+10])];
	}
	return;	
}

__device__ void min2(double *a, int n, int &min_idx)
{
	double min = a[0];
	for (int i = 0; i < n; i++)
	{
		if (min > a[i])
		{
			min = a[i];
			min_idx = i;
		}
	}
}

//PSO algorithm
__global__ void pso(double *param, double *MLAI, double *TMIN, double *TMAX, double *AVRAD, long  rand, double *gbest)
{
	hiprandState state;
	int idx = threadIdx.x+blockDim.x*blockIdx.x;	
	if(idx<K)
	{
	long seed = rand;
	hiprand_init(seed, idx, 0, &state);
	double mv = (param[3] - param[2]) / 2;	
	double MLAI1[NT];
	for (int s = 0; s < NT; s++)
	{
		MLAI1[s] = MLAI[idx*2 + s];
	}	
	double vel[PS];
	double pos[PS];
	double pbest[PS2];	
	double randnum1;
	double randnum2;
	for (int i = 0; i < PS; i++)
	{
		randnum1 = abs(hiprand_uniform_double(&state));
		vel[i] = 2*mv*randnum1 - mv;
		randnum2 = abs(hiprand_uniform_double(&state));
		pos[i] = (param[3] - param[2])*randnum2 + param[2];	
		if(i%2==0)
		{
		   pbest[i/2] = pos[i];
		}
	}
	double pbestval[PS2];
	double cost;
	double JCLAImoni[NT];
	for (int i = 0; i < PS; i++)
	{
		LAIcal(pos[i], param, TMIN, TMAX, AVRAD, JCLAImoni);		
		if(i%2==0)
		{
			cost=0;
			for (int j = 0; j < NT; j++)
			{			
				cost = cost + (MLAI1[j] - JCLAImoni[j])*(MLAI1[j] - JCLAImoni[j]);
			}
			pbestval[i/2] = sqrt(cost) / NT;		
		}
	}			
	int min_idx = 0;
	min2(pbestval, PS2, min_idx);
	double gbestval = pbestval[min_idx];
	gbest[idx] = pbest[min_idx];
	double cnt2 = 0;
	double iwt[EP] = { param[6] };	
    double pout[PS2];
	double randnum3;
	double randnum4;
	double tmp1=0;
	double tr=0;
	for (int j = 0; j < EP; j++)
	{
		for (int i = 0; i < PS; i++)
		{
			LAIcal(pos[i], param, TMIN, TMAX, AVRAD, JCLAImoni);							
			cost = 0;
			if(i%2==0)
			{
				for (int jj = 0; jj < NT; jj++)
				{
					cost = cost + (MLAI1[jj] - JCLAImoni[jj])*(MLAI1[jj] - JCLAImoni[jj]);
				}
				pout[i/2] = sqrt(cost) / NT;
			}
		}
		tr = gbestval;     
		for (int i = 0; i < PS2; i++)
		{
			if (pbestval[i] >= pout[i])
			{
				pbestval[i] = pout[i];
				pbest[i] = pos[2*i];
			}
		}
		int min_idx2;
		min2(pbestval, PS2, min_idx2);
		double iterbestval = pbestval[min_idx2];
		if (gbestval >= iterbestval)
		{
			gbestval = iterbestval;
			gbest[idx] = pbest[min_idx2];
		}      
		if (j <= EP)
			iwt[j] = ((param[7] - param[6]) / 14 )*(j - 1) + param[6];		    
		else
			iwt[j] = param[7];
		double ac11;
		double ac22;
		double pbest2[PS];		
		for (int i = 0; i < PS; i++)
		{			
			if(i%2==0)
			{
				pbest2[i]=pbest[i];
			}
			else
			{
				pbest2[i]=pbest[i-1];
			}
			randnum3=abs(hiprand_uniform_double(&state));
			randnum4=abs(hiprand_uniform_double(&state));
			ac11 = randnum3 * param[4];
			ac22 = randnum4 * param[5];
			vel[i] = iwt[i] * vel[i] + ac11 * (pbest2[i] - pos[i]) + ac22 * (gbest[idx] - pos[i]);
			vel[i] = LIMIT(-mv, mv, vel[i]);
			pos[i] = pos[i] + vel[i];
			pos[i] = LIMIT(param[2], param[3], pos[i]);
		}
		tmp1 = abs(tr - gbestval);
		if (tmp1 > param[8])
			cnt2 = 0;
		else
		{
			if (tmp1 <= param[8])
			{
				cnt2 = cnt2 + 1;
				if (cnt2 >= param[9])
					break;
			}
		}		
	}
	}
	
}


using namespace std;

//main function
int main()
{	
	srand((unsigned int)time(NULL));
	const int nop = NT + 10;
	double param[nop];
	param[0] = 154;                   
	param[1] = Y - param[0] + 1;      
	param[2] = 10;                   
	param[3] = 50;                  
	param[4] = 2.1;                 
	param[5] = 1.6;                 
	param[6] = 0.9;                 
	param[7] = 0.6;                 
	param[8] = 1e-99;               
	param[9] = 10;                  
    param[10] = 194;                
	param[11] = 210;                
	param[12] = 216;
	param[13] = 259;	
	int i, j;
	double** MLAI= new double* [K];
	for (i=0;i<K;i++)
	{
		MLAI[i]=new double [NT];		
	}	
	ifstream fin("MLAI_2015.txt");         
	for (i = 0; i < K; i++)
	{
	    for (j = 0; j<NT; j++)
		{
		     fin >> MLAI[i][j];
		}	    
	}
	fin.close();
	const int totalpix = K*NT;
	double *MLAI1=new double [totalpix];
	
	for (int p = 0; p < K; p++)
	{
		for (int q = 0; q < NT; q++)
		{
			MLAI1[p*NT+q] = MLAI[p][q];			
		}
	}
	double TMIN[Y];
	ifstream fin1("TMIN_2015.txt");       
	for (i = 0; i<Y; i++)
		fin1 >> TMIN[i];
	fin1.close();
	double TMAX[Y];
	ifstream fin2("TMAX_2015.txt");
	for (i = 0; i<Y; i++)
		fin2 >> TMAX[i];
	fin2.close();
	double AVRAD[Y];
	ifstream fin3("AVRAD_2015.txt");
	for (i = 0; i<Y; i++)
		fin3 >> AVRAD[i];
	fin3.close();
	for (int i = 0; i < Y; i++)
	{
		AVRAD[i] = AVRAD[i] * 1000;
	}
	double *gbest=new double [K];
	double *d_param;
	double *d_MLAI;
	double *d_TMIN;
	double *d_TMAX;
	double *d_AVRAD;
	double *d_gbest;	
	hipSetDevice(5);
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start, 0);	
	hipError_t cudastate;
	cudastate = hipMalloc((void**)&d_param, sizeof(double)* nop); CHECK(cudastate)
	cudastate = hipMalloc((void**)&d_MLAI, sizeof(double)*totalpix); CHECK(cudastate)
	cudastate = hipMalloc((void**)&d_TMIN, sizeof(double)*Y); CHECK(cudastate)
	cudastate = hipMalloc((void**)&d_TMAX, sizeof(double)*Y); CHECK(cudastate)
	cudastate = hipMalloc((void**)&d_AVRAD, sizeof(double)*Y); CHECK(cudastate)
	cudastate = hipMalloc((void**)&d_gbest, sizeof(double)*K); CHECK(cudastate)
	hipMemcpy(d_param, param, sizeof(double)* nop, hipMemcpyHostToDevice);
	hipMemcpy(d_MLAI, MLAI1, sizeof(double)*totalpix, hipMemcpyHostToDevice);
	hipMemcpy(d_TMIN, TMIN, sizeof(double)*Y, hipMemcpyHostToDevice);
	hipMemcpy(d_TMAX, TMAX, sizeof(double)*Y, hipMemcpyHostToDevice);
	hipMemcpy(d_AVRAD, AVRAD, sizeof(double)*Y, hipMemcpyHostToDevice);
	for(i=0; i<K; ++i) 
	{
		delete[] MLAI[i];
	}
	delete[] MLAI;
	delete[] MLAI1;	
	int  thread = 256;
	int  block = K / thread;
	dim3 dimGrid(block + 1);
	dim3 dimBlock(thread);
	pso <<<dimGrid, dimBlock >>>(d_param, d_MLAI, d_TMIN, d_TMAX, d_AVRAD,rand(), d_gbest);
    cudastate = hipDeviceSynchronize(); CHECK(cudastate)
	hipMemcpy(gbest, d_gbest, sizeof(double)*K, hipMemcpyDeviceToHost);	
	float GPU_time;
	hipEventRecord(end, 0);
	hipEventSynchronize(end); 		
	hipEventElapsedTime(&GPU_time, start, end);	
	cout << "parallel time："<< GPU_time/1000 << endl;	
	FILE *p = fopen("gbest_2015.txt", "wt");
	for (int i = 0; i<K; i++) 
		fprintf(p, "%4.2f\n", gbest[i]);
	fclose(p);	
	delete[] gbest;
	cudastate = hipFree(d_param); CHECK(cudastate)
	cudastate = hipFree(d_MLAI); CHECK(cudastate)
	cudastate = hipFree(d_TMIN); CHECK(cudastate)
	cudastate = hipFree(d_TMAX); CHECK(cudastate)
	cudastate = hipFree(d_AVRAD); CHECK(cudastate)
	cudastate = hipFree(d_gbest); CHECK(cudastate)
}

